#include "hip/hip_runtime.h"
// local
#include <los.h>
// ansi
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// cuda
#include <cutil.h>


//=====================================
// CONSTANTS, MACROS, DEFINITIONS
//=====================================

#define BLOCK_SIZE  16


//========================================
// DEVICE FUNCTIONS
//========================================

__device__ float _sampleBicubicFilter(float *hmap, unsigned int w, unsigned int h, float u, float v)
{
    unsigned int baseu, basev;
    float heights[6];

    // get the base index for the 2x2 block
    baseu = floorf(u);
    basev = floorf(v);

    // sample the 2x2 texel block
    heights[0] = hmap[basev*w+baseu];
    heights[1] = hmap[basev*w+min(baseu+1,w-1)];
    heights[2] = hmap[min(basev+1,h-1)*w+baseu];
    heights[3] = hmap[min(basev+1,h-1)*w+min(baseu+1,w-1)];

    // lerp along the horizontal (2 rows)
    heights[4] = heights[0] + (heights[1] - heights[0]) * (u - baseu);  // lerp 0 & 1
    heights[5] = heights[2] + (heights[3] - heights[2]) * (u - baseu);  // lerp 2 & 3
    return heights[4] + (heights[5] - heights[4]) * (v - basev);        // lerp results above
}

__global__ void _losPointRectDEVICE(unsigned char *los, float *hmap, unsigned int w, unsigned int h, unsigned int pt)
{
    int x  = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    int y  = BLOCK_SIZE * blockIdx.y + threadIdx.y;
    int bi = y * w + x;
    float ax,ay,az;
    float bx,by,bz;
    float cx,cy,cz;
    float length, sample;
    float i,t;

    // point A
    ax = pt % w;
    ay = pt / w;
    az = hmap[pt];

    // point B
    bx = x;
    by = y;
    bz = hmap[bi];

    // line AB
    length = sqrtf( ((bx-ax)*(bx-ax)) + ((by-ay)*(by-ay)) + ((bz-az)*(bz-az)) );

    // los algorithm: sample hmap along line
    i=1.0f; t=0;
    while (i<length)
    {
        t = i / length;
        cx = (ax*(1.0f-t)) + (bx*t);
        cy = (ay*(1.0f-t)) + (by*t);
        cz = (az*(1.0f-t)) + (bz*t);
        sample = _sampleBicubicFilter(hmap,w,h,cx,cy);

        if (sample-cz>1.0f) // use function 'signbit()'
        {
            los[bi]=LOS_BLOCKED;
            break;
        }

        i+=0.5f;
    }     
}


//=====================================
// HOST FUNCTIONS
//=====================================

__host__ void initDevice()
{
    CUT_DEVICE_INIT();
}

__host__ void killDevice(int argc, const char *argv)
{
    CUT_EXIT(argc, argv);
}

__host__ void timerCreate(unsigned int *timer)
{
    CUT_SAFE_CALL(cutCreateTimer(timer));
}

__host__ void timerStart(unsigned int timer)
{
    CUT_SAFE_CALL(cutResetTimer(timer));
    CUT_SAFE_CALL(cutStartTimer(timer));
}

__host__ float timerStop(unsigned int timer)
{
    CUT_SAFE_CALL(cutStopTimer(timer));
    return (cutGetTimerValue(timer) / 1000.0f);
}

__host__ void losPointRectDEVICE(unsigned char *los, float *hmap, unsigned int w, unsigned int h, unsigned int pt)
{
    unsigned int HMAP_SIZE = w*h;
    unsigned char  *d_los;  CUDA_SAFE_CALL(hipMalloc((void**)&d_los,  HMAP_SIZE*sizeof(unsigned char)));
    float          *d_hmap; CUDA_SAFE_CALL(hipMalloc((void**)&d_hmap, HMAP_SIZE*sizeof(float)));

    CUDA_SAFE_CALL(hipMemcpy(d_los,  los,  HMAP_SIZE*sizeof(unsigned char), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_hmap, hmap, HMAP_SIZE*sizeof(float),         hipMemcpyHostToDevice));    
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(w/BLOCK_SIZE, h/BLOCK_SIZE);
    _losPointRectDEVICE<<<blocks, threads>>>(d_los, d_hmap, w, h, pt);
    CUT_CHECK_ERROR("Kernel execution failed!");
    CUDA_SAFE_CALL(hipMemcpy(los, d_los, HMAP_SIZE*sizeof(unsigned char), hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipFree(d_los));
    CUDA_SAFE_CALL(hipFree(d_hmap));
}
